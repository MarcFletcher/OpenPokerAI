
#include <hip/hip_runtime.h>
//THIS IS CURRENTLY BROKEN DO NOT USE!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!


/*
 * Copyright 1993-2009 NVIDIA Corporation.  All rights reserved.
 *
 * NVIDIA Corporation and its licensors retain all intellectual property and 
 * proprietary rights in and to this software and related documentation and 
 * any modifications thereto.  Any use, reproduction, disclosure, or distribution 
 * of this software and related documentation without an express license 
 * agreement from NVIDIA Corporation is strictly prohibited.
 * 
 */

#define  DCMT_SEED 4172
#define  MT_RNG_PERIOD 607


typedef struct{
    unsigned int matrix_a;
    unsigned int mask_b;
    unsigned int mask_c;
    unsigned int seed;
} mt_struct_stripped;


#define   MT_RNG_COUNT 4096
#define   MT_MM 9
#define   MT_NN 19
#define   MT_WMASK 0xFFFFFFFFU
#define   MT_UMASK 0xFFFFFFFEU
#define   MT_LMASK 0x1U
#define   MT_SHIFT0 12
#define   MT_SHIFTB 7
#define   MT_SHIFTC 15
#define   MT_SHIFT1 18

//__device__ static mt_struct_stripped ds_MT[MT_RNG_COUNT];

////////////////////////////////////////////////////////////////////////////////
// Write MT_RNG_COUNT vertical lanes of NPerRng random numbers to *d_Random.
// For coalesced global writes MT_RNG_COUNT should be a multiple of warp size.
// Initial states for each generator are the same, since the states are
// initialized from the global seed. In order to improve distribution properties
// on small NPerRng supply dedicated (local) seed to each twister.
// The local seeds, in their turn, can be extracted from global seed
// by means of any simple random number generator, like LCG.
////////////////////////////////////////////////////////////////////////////////
extern "C" __global__ void RandomGPU(
    float *d_Random,    
	char *ds_MT_Bytes,
	unsigned int *seeds,
	int NPerRng	
){
    const int      tid = blockDim.x * blockIdx.x + threadIdx.x;   
	const int seedCount = (blockDim.x * (blockIdx.x + 1)) / MT_RNG_COUNT;
	
	mt_struct_stripped *ds_MT = (mt_struct_stripped*)ds_MT_Bytes;
	mt_struct_stripped config = ds_MT[tid - seedCount * MT_RNG_COUNT];
	
	unsigned int matrix_a = config.matrix_a;
	unsigned int mask_b = config.mask_b;
	unsigned int mask_c = config.mask_c;

    int iState, iState1, iStateM, iOut;
    unsigned int mti, mti1, mtiM, x;
    unsigned int mt[MT_NN];

    
    mt[0] = seeds[seedCount];
    for(iState = 1; iState < MT_NN; iState++)
        mt[iState] = (1812433253U * (mt[iState - 1] ^ (mt[iState - 1] >> 30)) + iState) & MT_WMASK;

    iState = 0;
    mti1 = mt[0];
    for(iOut = 0; iOut < NPerRng; iOut++){
        iState1 = iState + 1;
        iStateM = iState + MT_MM;
        if(iState1 >= MT_NN) iState1 -= MT_NN;
        if(iStateM >= MT_NN) iStateM -= MT_NN;
        mti  = mti1;
        mti1 = mt[iState1];
        mtiM = mt[iStateM];

        x    = (mti & MT_UMASK) | (mti1 & MT_LMASK);
        x    =  mtiM ^ (x >> 1) ^ ((x & 1) ? matrix_a : 0);
        mt[iState] = x;
        iState = iState1;

        
        x ^= (x >> MT_SHIFT0);
        x ^= (x << MT_SHIFTB) & mask_b;
        x ^= (x << MT_SHIFTC) & mask_c;
        x ^= (x >> MT_SHIFT1);
        
        d_Random[tid + iOut * gridDim.x * blockDim.x] = ((float)x + 1.0f) / 4294967296.0f;		
	}
}
